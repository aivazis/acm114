// -*- C++ -*-
// -*- coding: utf-8 -*-
//
// ~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~
//
//                             michael a.g. aïvázis
//                      california institute of technology
//                      (c) 1998-2010  all rights reserved
//
// ~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~
//

#include <getopt.h> // for getopt and friends
#include <hip/hip_runtime.h>

#include <cmath>
#include <iostream>


// each thread computes 
__global__ void dilog_cuda(float* array, float zp, long workers, long subdivisions) {
    // compute its offset using the block geometry builtins
    int idx = blockIdx.x * blockDim.x  + threadIdx.x;
    // make sure we don't go past the last one
    if (idx < workers && idx < subdivisions) {
        float sum = 0.0f;
        long load = subdivisions/workers;

        float dz = zp/subdivisions;
        float z = (idx-1.f)*load*dz + dz/2;

        for (int i=0; i<load; i++) {
            sum += log(1-z)/z;
            z += dz;
        }
        // do the arithmetic
        array[idx] = sum;
    }
    return;
}

int main(int argc, char* argv[]) {
    //  default values for the command line options
    long N = 1000;
    double z = 1.0;
    // read the command line
    int command;
    while ((command = getopt(argc, argv, "z:N:")) != -1) {
        switch (command) {
        case 'z':
            // get the argument of the dilogarithm 
            z = atof(optarg);
            break;
        case 'N':
            // get the number of subdivisions
            N = (long) atof(optarg);
            break;
        }
    }

    // geometry
    int warp = 32;
    int cores = 8;
    int processors = 30;
    size_t workers = processors*cores*warp;
    // allocate a block on the host
    float* partials_host = new float[workers];
    for (int i=0; i<workers; i++) {
        partials_host[i] = 0.f;
    }
    // allocate a block on the device
    float* partials_dev;
    hipMalloc((void **) &partials_dev, workers*sizeof(float));
    // send the data from the host to the device
    hipMemcpy(partials_dev, partials_host, workers*sizeof(float), hipMemcpyHostToDevice);

    // set up the device execution context for our threads
    // each thread will take care of one element
    int blockSz = warp; // a warp at a time
    // compute the number of blocks needed
    int nBlocks = cores*processors;
    // compute the load per thread
    long load = N/workers + 1;
    N = load*workers;
    float dz = z/N;
    // compute
    dilog_cuda <<<nBlocks, blockSz>>> (partials_dev, z, (long)workers, N);
    // get the partials back on the host
    hipMemcpy(partials_host, partials_dev, workers*sizeof(float), hipMemcpyDeviceToHost);

    // do the reduction
    double sum = 0.f;
    for (int i=0; i<workers; i++) {
        sum += partials_host[i];
    }
    sum *= -dz;

    std::cout << "sum = " << sum << std::endl;
    // all done
    return 0;
}

// end of file
