// -*- C++ -*-
// -*- coding: utf-8 -*-
//
// ~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~
//
//                             michael a.g. aïvázis
//                      california institute of technology
//                      (c) 1998-2010  all rights reserved
//
// ~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~
//

// memxchng.cu
#include <hip/hip_runtime.h>
#include <assert.h>

int main(int argc, char* argv[]) {
    const int N = 12;
    // allocate some buffers on the host
    float *send_host = (float *) malloc(N*sizeof(float));
    float *recv_host = (float *) malloc(N*sizeof(float));
    // allocate matching ones on the device
    float *send_device, *recv_device;
    hipMalloc((void **) &recv_device, N*sizeof(float));
    hipMalloc((void **) &send_device, N*sizeof(float));
    // and initialize the host data
    for (int i=0; i<N; i++) {
        send_host[i] = 2.0f + i*i;
        recv_host[i] = 0.0f;
    }
    // send the data from the host to the device
    hipMemcpy(recv_device, send_host, N*sizeof(float), hipMemcpyHostToDevice);
    // move the data in device memory
    hipMemcpy(send_device, recv_device, N*sizeof(float), hipMemcpyDeviceToDevice);
    // get it back on the host
    hipMemcpy(recv_host, send_device, N*sizeof(float), hipMemcpyDeviceToHost);
    // check the result
    for (int i=0; i<N; i++) {
      assert(send_host[i] == recv_host[i]);
    }
    // free the buffers;
    free(send_host); free(recv_host);
    hipFree(send_device); hipFree(recv_device);

    return 0;
}

// end of file
