// -*- C++ -*-
// -*- coding: utf-8 -*-
//
// ~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~
//
//                             michael a.g. aïvázis
//                      california institute of technology
//                      (c) 1998-2010  all rights reserved
//
// ~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~
//

// memxchng.cu
#include <hip/hip_runtime.h>
#include <assert.h>


// manipulate the host array
void scale_host(float* array, float scale, int N) {
    // loop over all array elements and multiply them by 2 
    for (int idx=0; idx<N; idx++) {
        array[idx] *= scale;
    }
    return;
}

// and here is the corresponding code for the GPU
__global__ void scale_dev(float* array, float scale, int N) {
    // this thread is responsible for one element of the array
    // compute its offset using the block geometry builtins
    int idx = blockIdx.x * blockDim.x  + threadIdx.x;
    // make sure we don't do go past the last one
    if (idx < N) {
        // do the arithmetic
        array[idx] *= scale;
    }
    return;
}

int main(int argc, char* argv[]) {
    const int N = 12;
    // allocate some buffers on the host
    float* send_host = (float *) malloc(N*sizeof(float));
    float* recv_host = (float *) malloc(N*sizeof(float));
    // allocate matching ones on the device
    float* array_dev;
    hipMalloc((void **) &array_dev, N*sizeof(float));

    // and initialize the host data
    for (int i=0; i<N; i++) {
        send_host[i] = 2.0f + i*i;
        recv_host[i] = 0.0f;
    }

    // send the data from the host to the device
    hipMemcpy(array_dev, send_host, N*sizeof(float), hipMemcpyHostToDevice);

    // set up the device execution context for our threads
    // each thread will take care of one element
    int blockSz = 4; // 4 threads per block
    // compute the number of blocks needed
    int nBlocks = N/blockSz; 
    // adjust up to make sure we cover the entire array
    if (N % nBlocks) {
        nBlocks++;
    }
    // scale the array on the device
    float scale = 2.0f;
    scale_dev <<<nBlocks, blockSz>>> (array_dev, scale, N);
    // scale the input array on the host
    scale_host(send_host, scale, N);

    // get it back on the host
    hipMemcpy(recv_host, array_dev, N*sizeof(float), hipMemcpyDeviceToHost);

    // check the result
    for (int i=0; i<N; i++) {
      assert(send_host[i] == recv_host[i]);
    }
    // free the buffers;
    hipFree(array_dev);
    free(send_host); free(recv_host);

    return 0;
}

// end of file
